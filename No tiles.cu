#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

__global__ void matrix_mult_kernel(int *M, int *N, int *P, int RM, int CM, int CN)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (row < RM && col < CN) {
        int end_res= 0;
        for (int i = 0; i < cM; i++) {
            end_res += M[row*RM + i] * N[i*CM + col];
        }
        P[row*CN + col] = end_res;
    }
}
int main()
{
      int RM;
      int CM;
      int CN;
      scanf("%d %d %d", &RM, &CM, &CN);

      int s1 = RM*CM;
      int s2 = CM*CN;
      int s3 = RM*CN;
      
      int *M = (int *)malloc(sizeof(int)*s1);
      int *N = (int *)malloc(sizeof(int)*s2);
      int *p = (int *)malloc(sizeof(int)*s3);

      for(int i=0;i<RM;i++)
      {
        for(int j=0;j<CM;j++)
        {
          M[i*RM+j]=rand()%1000;
        }
      }
      for(int i=0;i<colM;i++)
      {
        for(int j=0;j<colN;j++)
        {
          N[i*CM+j]=rand()%1000;
        }
      }

    int *d_M, *d_N, *d_p;
    hipMalloc(&d_M, sizeof(int)*s1);
    hipMalloc(&d_N, sizeof(int)*s2);
    hipMalloc(&d_p, sizeof(int)*s3);

    hipMemcpy(d_M, M, sizeof(int)*size1, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, sizeof(int)*size2, hipMemcpyHostToDevice);

    
    dim3 gridDim((CN + blockDim.x - 1) / blockDim.x, (RM + blockDim.y - 1) / blockDim.y);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrix_mult_kernel<<<gridDim, blockDim>>>(d_M, d_N, d_p, RM, CM, CN);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(p, d_p, sizeof(int)*s3, hipMemcpyDeviceToHost);
       
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_p);
    
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time of execution: %.8f milliseconds\n", milliseconds);

    return 0;
    
  }