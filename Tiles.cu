#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void tiles_matrix_mult(int *M, int *N, int *P,int RM, int CM, int CN)
{
    __shared__ int m[16][16];
    __shared__ int n[16][16];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    int res= 0;

    for (int i = 0; i < (CN - 1) / 16 + 1; i++)
    {
        if (row < rowM && tx * 16 + tx < CM)
        {
            m[ty][tx] = M[row * CM + i * 16 + tx];
        }
        else
        {
            m[ty][tx] = 0;
        }

        if (i * 16 + ty < CM && col < CN)
        {
            n[ty][tx] = N[(i * 16 + ty) * CN + col];
        }
        else
        {
            n[ty][tx] = 0;
        }

        __syncthreads();

        if (row < RM && col < CN)
        {
            for (int j = 0; j < 16; j++)
            {
                prod += m[ty][j] * n[j][tx];
                __syncthreads();
            }
        }
    }

    if (row < RM && col < CN)
    {
        P[row * colN + col] = res
    }
}

int main()
{
    

    int RM, CM, CN;
    scanf("%d %d %d", &RM, &CM, &CN);

    int size1= RM*CM;
    int size2 = CM* CN;
    int size3 = RM*CN;

    int *M = (int *)malloc(sizeof(int) * size1);
    int *N = (int *)malloc(sizeof(int) * size2);
    int *P = (int *)malloc(sizeof(int) * size3);

    for (int i = 0; i < RM; i++)
    {
        for (int j = 0; j < CM; j++)
        {
            M[i * RM + j] = rand() % 1000 + 1;
        }
    }

    for (int i = 0; i < CM;i++)
    {
        for (int j = 0; j < CN;j++)
        {
            N[i * CM + j] = rand() % 1000 + 1;
        }
    }

    int *d_M, *d_N, *d_P;
    hipMalloc(&d_M, sizeof(int) * size1);
    hipMalloc(&d_N, sizeof(int) * size2);
    hipMalloc(&d_P, sizeof(int) * size3);

    hipMemcpy(d_M, M, sizeof(int) * size1, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, sizeof(int) * size2, hipMemcpyHostToDevice);

    dim3 blockDim(16,16);
    dim3 gridDim((CN + blockDim.x - 1) / blockDim.x, (RM + blockDim.y - 1) / blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    tiles_matrix_mult<<<gridDim, blockDim>>>(d_M, d_N, d_P, RM, CM, CN);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(P,d_P, sizeof(int) * size3, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time of execution: %.8f milliseconds\n", milliseconds);

    free(M);
    free(N);
    free(out);

    return 0;
}